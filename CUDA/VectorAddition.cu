#include "hip/hip_runtime.h"

#include<stdio.h>

#define SIZE 1024

__global__ void VectorAddition(int*a, int*b, int*c)
{
    int i = threadIdx.x;

    if (i < SIZE)
    {
        c[i] = a[i] + b[i];
    }
}

int main()
{
    int* MatrixA, *MatrixB, *MatrixC;

    hipMallocManaged(&MatrixA, SIZE * sizeof(int));
    hipMallocManaged(&MatrixB, SIZE * sizeof(int));
    hipMallocManaged(&MatrixC, SIZE * sizeof(int));

    hipDeviceSynchronize();

    for (int i = 0; i < SIZE; ++i)
    {
        MatrixA[i] = (2 * i);
        MatrixB[i] = ((2 * i) + 1);
        MatrixC[i] = 0;
    }

    VectorAddition<<<1, SIZE>>> (MatrixA, MatrixB, MatrixC);

    for (int i = 0; i < 10; i++)
    {
        printf("c[&d] = %d\n");
    }

    hipFree(MatrixA);
    hipFree(MatrixB);
    hipFree(MatrixC);
}
